
// nvcc -o cda 3loop_cuda.cu


#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

// Define matrix size
#define N 1024

// CUDA kernel to perform matrix multiplication
__global__ void matrixMult(int *d_A, int *d_B, int *d_C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; ++k) {
            sum += d_A[row * n + k] * d_B[k * n + col];
        }
        d_C[row * n + col] = sum;
    }
}

// Function to create matrix on host
void createMx(int *mx, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            mx[i * n + j] = i + j;
        }
    }
}

void printMx(int *mx, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            cout << mx[i * n + j] << " ";
        }
        cout << "\n";
    }
}

int main() {
    const int size = N * N * sizeof(int);
    float totalTime = 0.0f;

    // Allocate host memory for matrices
    int *h_A = new int[N * N];
    int *h_B = new int[N * N];
    int *h_C = new int[N * N];

    // Allocate device memory for matrices
    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Initialize matrices on host
    createMx(h_A, N);
    createMx(h_B, N);

    for (int i = 0; i < 10; i++) {
        // Define grid and block dimensions
        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

        // Copy data from host to device
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Create CUDA events for timing
        hipEvent_t start, stop;
        float elapsedTime;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Record start time
        hipEventRecord(start, 0);

        // Launch CUDA kernel to multiply matrices
        matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Record stop time
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        // Calculate elapsed time
        hipEventElapsedTime(&elapsedTime, start, stop);
        totalTime += elapsedTime;

        // Destroy CUDA events
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Calculate average time
    float avgTime = totalTime / 10.0f;

    cout << "\nTime taken for 3loop Matrix multiplication (CUDA): " 
         << avgTime << " milliseconds (" 
         << avgTime / 1000.0f << " seconds)" << endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    //printMx(h_C, 1024);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}